#include <stdio.h>
#include <hip/hip_runtime.h>


#define NUM 10

__global__ void averagePx(int * a, int * c)
{
    int width = blockIdx.x;

    printf("w: %d\n", width);

    if(width < NUM*NUM) {
        c[width] = a[width] / 2;
    }

}

int main ()
{
    int a[NUM][NUM], c[NUM*NUM];
    int change_a[NUM*NUM];
    int *de_a, *de_c;

    // 이미지의 값을 넣어 준다. 지금은 내 임의로 넣는다.
    int gap = 0;
    for (int i = 0; i < NUM; ++i) {
        for (int j = 0; j < NUM; ++j) {
            a[i][j] = 5 * 10;
            change_a[gap] = a[i][j];
            printf("a[%d][%d] = %d\n", i, j, a[i][j]);
            printf("gap[%d] = %d\n", gap, change_a[gap]);
            ++gap;
        }
    }

    // 이미지를 일차원으로 바꿔준다.


    hipMalloc((void**)&de_a, sizeof(int) * NUM*NUM);
    hipMalloc((void**)&de_c, sizeof(int) * NUM*NUM);

    hipMemcpy(de_a, change_a, sizeof(int) * NUM*NUM, hipMemcpyHostToDevice);

    averagePx <<<100, 1>>> (de_a, de_c);

    hipMemcpy(c, de_c, sizeof(int) * NUM*NUM, hipMemcpyDeviceToHost);

    for (int i = 0; i < NUM*NUM; ++i) {
            printf("a(%d) = c(%d)\n", change_a[i], c[i]);
    }

    hipFree(de_a);
    hipFree(de_c);

    getchar();
    return 0;

}